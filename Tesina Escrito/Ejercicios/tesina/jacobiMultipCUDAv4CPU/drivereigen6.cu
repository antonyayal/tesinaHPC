/* Driver for routine EIGSRT  */
#include <stdio.h>
//#include "nr.h"
//#include "nrutil.h"
#include <sys/time.h>
#include <stdlib.h>
//#include"jacobiMultipCUDA.h"

//#define NP 128

int main(int argc, char **argv)
{
    int NP;
    char *nombreArchivo=argv[1];
    double S,E;
	int i, j, nrot=0;
    FILE *archivo;
    float *c;

    if (fopen(nombreArchivo, "r") == NULL){
        printf("File not found\n");
        return 1;
    }else{
        archivo = fopen(nombreArchivo, "r");
        fscanf(archivo, "%d", &NP);
        c =(float *)matrix(1,NP-1,1,NP-1);
        for (i = 0; i < NP; i++){
            for (j = 0; j < NP; j++){
                fscanf(archivo, "%f", &c[i*NP+j]);
            }
        }
        fclose(archivo);
    }
        float *d, *v, *e;
        
        d=(float *)vector(1,NP);
        v=(float *)matrix(1,NP,1,NP);
        e=(float *)convert_matrix(c,1,NP,1,NP);
        
        for (i = 1; i <= NP; ++i){
            printf("\n");
            for (j = 1; j <= NP; ++j)
                printf("[%f]",e[i*NP+j]);
        }

        printf("\n****** Finding Eigenvectors ******\n");
        //jacobi(e,NP,d,v,&nrot);
        get_walltime(&S);
              
        jacobiMultip(e,NP,NP,v,d,&nrot);
        
        get_walltime(&E);

        for (i = 1; i <= NP; ++i){
            printf("\n");
            for (j = 1; j <= NP; ++j)
                printf("[%f]",v[i*NP+j]);
        }
        
        printf("\nd\n");
        for (i = 1; i <= NP; ++i)
        {
            printf("[%f]",d[i]);
        }

        printf("\n******  Eigenvalues & Eigenvectors ******\n");
        for (i=1;i<=NP; i++) {
            printf("eigenvalue %3d, = %12.6f\n",i,d[i]);
            printf("eigenvector:\n");
            for (j=1;j<=NP; j++) {
                printf("%12.6f",v[i*NP+j]);
                if ((j % 5) == 0) printf("\n");
            }
                printf("\n");
        }

    printf("Rotations: %d\n",nrot );

    printf("Total time:%f sec\n", E-S);
	

    //free_vector(d,1,NP);
    //free_vector(v,1,NP*NP);
    //free_vector(e,1,NP*NP);
    // free(d);
    // free(v);
    // free(e);

    return 0;
	
}
