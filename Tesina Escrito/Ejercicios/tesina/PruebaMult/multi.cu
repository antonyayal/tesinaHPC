
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

__global__ void kernel_mat_mult(int *n,float *A, float *B,float *C){

	float Cvalue;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int e;

	if(row > (*n)|| col > (*n)) 
		return;

	for (row = row; row < (*n); ++row){
		Cvalue = 0.0;
		for (e = 0; e < (*n); e++)
			Cvalue += (A[row * (*n) + e]) * (B[e * (*n) + col]);
		C[row * (*n) + col] = Cvalue;
	}
}

__global__ void imprimirMatDev(int *n,float *A,char ch){
	int i,j;
	printf("\nMatrix %c\n",ch);
	for (i = 0; i < *n; ++i){
            printf("\n");
            for (j = 0; j < *n; ++j)
                printf("[%f]",A[i*(*n)+j]);
    }
}

int main(int argc, char const *argv[])
{
	int n=3;
	int *d_n;
	float *d_mat_A;
	float *d_mat_B;
	float *d_mat_C;
	size_t size = (n+1) * (n+1) * sizeof(float);
	dim3 dimGrid( 32 );         // 512 x 1 x 1
	dim3 dimBlock( 64); // 1024 x 1024 x 1 

	float mat_A[3*3]= 
        {2.0,-1.0,0.0,
        -1.0,2.0,-1.0,
         0.0,-1.0,2.0,};
    float mat_B[3*3]= 
        {2.0,0.0,0.0,
         0.0,2.0,0.0,
         0.0,0.0,2.0,};
float mat_C[3*3]= 
        {0.0,0.0,0.0,
         0.0,0.0,0.0,
         0.0,0.0,0.0,};
	hipMalloc(&d_n,sizeof (int));
	hipMalloc(&d_mat_A, size);
	hipMalloc(&d_mat_B, size);
	hipMalloc(&d_mat_C, size);


	hipMemcpy(d_n, &n, sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat_A, mat_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mat_B, mat_B, size, hipMemcpyHostToDevice);
	imprimirMatDev<<<1,1>>>(d_n,d_mat_A,'A');
	printf("\n");
	imprimirMatDev<<<1,1>>>(d_n,d_mat_B,'B');
	printf("\n");
	kernel_mat_mult<<<dimGrid, dimBlock>>>(d_n,d_mat_A,d_mat_B,d_mat_C);
	imprimirMatDev<<<1,1>>>(d_n,d_mat_C,'C');
	printf("\n");
	hipMemcpy(mat_C, d_mat_C, size, hipMemcpyDeviceToHost);

	return 0;
}