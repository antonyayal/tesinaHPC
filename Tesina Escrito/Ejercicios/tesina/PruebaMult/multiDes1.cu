
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

__global__ void kernel_mat_mult(int *n,float *A, float *B,float *C){

	float Cvalue;
	int row = blockIdx.y * blockDim.y + threadIdx.y+1;
	int col = blockIdx.x * blockDim.x + threadIdx.x+1;
	int e;

	if(row > (*n)|| col > (*n)) 
		return;

	for (row = row; row < (*n); ++row){
		Cvalue = 0.0;
		for (e = 1; e < (*n); e++)
			Cvalue += (A[row * (*n) + e]) * (B[e * (*n) + col]);
		C[row * (*n) + col] = Cvalue;
	}
}

__global__ void kernel_mat_mult_tras(int *n,float *A, float *B,float *C){

	float Cvalue;
	int row = blockIdx.y * blockDim.y + threadIdx.y+1;
	int col = blockIdx.x * blockDim.x + threadIdx.x+1;
	int e,i;

	if(row > (*n)|| col > (*n)) 
		return;

	for (i = col; i < (*n); ++i){
		Cvalue = 0.0;
		for (e = 1; e < (*n); e++)
			Cvalue += (A[e * (*n) + i]) * (B[e * (*n) + col]);
		C[i * (*n) + col] = Cvalue;
	}

	// for (col = col; col < (*n); ++col,++row){
	// 	Cvalue = 0.0;
	// 	for (e = 1; e < (*n); e++)
	// 		Cvalue += (A[e * (*n) + col]) * (B[e * (*n) + col]);
	// 	C[row * (*n) + col] = Cvalue;
	// }
}

__global__ void imprimirMatDev(int *n,float *A,char ch){
	int i,j;
	printf("\nMatrix %c\n",ch);

	for (i = 1; i < *n; ++i){
            printf("\n");
            for (j = 1; j < *n; ++j)
                printf("[%f]",A[i*(*n)+j]);
    }
}

int main(int argc, char const *argv[])
{
	int n=4;
	int *d_n;
	float *d_mat_A;
	float *d_mat_B;
	float *d_mat_C;
	size_t size = (n+1) * (n+1) * sizeof(float);
	dim3 dimGrid( 32 );         // 512 x 1 x 1
	dim3 dimBlock( 64); // 1024 x 1024 x 1 

	float mat_A[4*4]= 
        {0.0,0.0,0.0,0.0,
         0.0,2.0,-1.0,0.0,
        0.0,-1.0,2.0,-1.0,
         0.0,0.0,-1.0,2.0,};
    float mat_B[4*4]= 
        {0.0,0.0,0.0,0.0,
         0.0,1.0,0.0,0.0,
         0.0,0.0,1.0,0.0,
         0.0,0.0,0.0,1.0,};
float mat_C[4*4]= 
        {0.0,0.0,0.0,0.0,
         0.0,0.0,0.0,0.0,
         0.0,0.0,0.0,0.0,
         0.0,0.0,0.0,0.0,};
	hipMalloc(&d_n,sizeof (int));
	hipMalloc(&d_mat_A, size);
	hipMalloc(&d_mat_B, size);
	hipMalloc(&d_mat_C, size);

	printf("[%f]\n",mat_A[5] );
	printf("[%f]\n",mat_A[1*3+1] );

	hipMemcpy(d_n, &n, sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat_A, mat_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mat_B, mat_B, size, hipMemcpyHostToDevice);
	imprimirMatDev<<<1,1>>>(d_n,d_mat_A,'A');
	printf("\n");
	imprimirMatDev<<<1,1>>>(d_n,d_mat_B,'B');
	printf("\n");
	kernel_mat_mult<<<dimGrid, dimBlock>>>(d_n,d_mat_A,d_mat_B,d_mat_C);
	imprimirMatDev<<<1,1>>>(d_n,d_mat_C,'C');
	printf("\n");
	kernel_mat_mult_tras<<<dimGrid, dimBlock>>>(d_n,d_mat_A,d_mat_B,d_mat_C);
	imprimirMatDev<<<1,1>>>(d_n,d_mat_C,'D');
	hipMemcpy(mat_C, d_mat_C, size, hipMemcpyDeviceToHost);

	return 0;
}