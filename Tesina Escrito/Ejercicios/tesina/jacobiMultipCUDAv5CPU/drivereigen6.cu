/* Driver for routine EIGSRT  */

#include <hip/hip_runtime.h>
#include <stdio.h>
//#include "nr.h"
//#include "nrutil.h"
#include <sys/time.h>
#include <stdlib.h>
//#include"jacobiMultipCUDA.h"

//#define NP 128



#include <stdbool.h>
#define NR_END 1
#define FREE_ARG char*

void get_walltime_(double* wcTime) {
   struct timeval tp;
   gettimeofday(&tp, NULL);
   *wcTime = (double)(tp.tv_sec + tp.tv_usec/1000000.0);
}

void get_walltime(double* wcTime) {
   get_walltime_(wcTime);
}

void nrerror(char error_text[])
/* Numerical Recipes standard error handler */
{
	fprintf(stderr,"Numerical Recipes run-time error...\n");
	fprintf(stderr,"%s\n",error_text);
	fprintf(stderr,"...now exiting to system...\n");
	exit(1);
}

float *vector(long nl, long nh)
/* allocate a float vector with subscript range v[nl..nh] */
{
	float *v;

	v=(float *)malloc((size_t) ((nh-nl+1+NR_END)*sizeof(float)));
	if (!v) nrerror("allocation failure in vector()");
	return v-nl+NR_END;
}

float *matrix(long nrl, long nrh, long ncl, long nch)
/* allocate a float matrix with subscript range m[nrl..nrh][ncl..nch] */
{
	long N,
        nrow=nrh-nrl+1, 
        ncol=nch-ncl+1;
    N = (nrow+1)*(ncol+1);
    float *m;

    m=(float *) malloc((size_t)(N*sizeof(float)));
	if (!m) nrerror("allocation failure in matrix()");
    return m;
}

float* convert_matrix( float *a, long nrl, long nrh, long ncl, long nch)
{
	/* admpffpsjfpsdjpfsfpffssfpdofdpofdofsop */

    long i,j,k,N,
        nrow=nrh-nrl+1, ncol=nch-ncl+1;
    N = (nrow+1)*(ncol+1);
    float *m;

    m=(float *) malloc((size_t)(N*sizeof(float)));

    for (i = 1,k=0; i <= nrow; ++i)
        for (j = 1; j <= ncol; ++j,k++)
        	m[i*(ncol)+j]=a[k];

    if (!m) nrerror("allocation failure in convert_matrix()");
	 
    // for (i = 1; i <= nrow; ++i){
    //     printf("\n");
    //     for (j = 1; j <= ncol; ++j)
    //         printf("[%f]",m[i*ncol+j]);
    // }      
    return m;
    
}

void free_vector(float *v,long nl,long nh){
/* free a float vector allocated with vector() */

    free((FREE_ARG) (v+nl-NR_END));
}

//void jacobiMultip (float *mat, int n, int ndm, float *eigvec, float eigval[],  int *nrot);







void max_elem(int *piv_elem,int n,float *mat){
	int r,c;
	int max_i = 1;											//first coordenate i
	int max_j = 2;											//first coordenate j

	//#pragma acc loop 
	for (r = 1; r <= n-1; r++) 
      	for (c = r+1; c <= n; c++)
      		if(fabs(mat[r*n+c]) > fabs(mat[max_i*n+max_j])){ //if exists a higher element
      			max_i = r;									//replace new coor
      			max_j = c;
    		}
    piv_elem[0] = max_i;									//store new coordenates
    piv_elem[1] = max_j;

}

float cal_tan(int max_i,int max_j,float *mat, int n){ 
	float num;
	float den;
	float a1;
	float a2;
	float a3;

	num = 2 * (mat[max_i*n+max_j]);								
	if(mat[max_i*n+max_i] < mat[max_i*n+max_i])
		num = -num;

	a1 = mat[max_i*n+max_i] - mat[max_j*n+max_j]; 
	a2 = a1*a1;
	a3 = 4 * mat[max_i*n+max_j]*mat[max_i*n+max_j];
	den = a2 + a3;
	den = sqrt(den);
	den = abs(a1) + den;
	return num/den;
}

float cal_cos(float tang){						//cos = 1/√(1+tan^2)
	float cose;
	cose = 1 + (tang * tang);
	cose = sqrt(cose);
	cose = 1 / cose;
	return cose;
}

float cal_sin(float cose, float tang){			//sin = cos*tan
	float sino;
	sino = cose*tang;
	return sino;
}

void mat_mult(int n,float *mat, float *T,float *mat_temp){
	int i,j,k;

	//#pragma acc loop
	for (i = 1 ; i <= n ; i++ ){ 				//Premultiplication
    	for (j = 1 ; j <= n ; j++ ){
      		mat_temp[i*n+j] = 0;
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            mat_temp[i*n+j] += T[k*n+i] * mat[k*n+j];
			}
		}
	}

	//#pragma acc loop
	for (i = 1 ; i <= n ; i++ ){					//Postmultiplication
    	for (j = 1 ; j <= n ; j++ ){
      		mat[i*n+j] = 0;
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            mat[i*n+j] += mat_temp[i*n+k] * T[k*n+j];
			}
		}
	}
}

void mat_mult2(int n,float *A, float *B,float *C){
	int i,j,k;
	for (i = 1 ; i <= n ; i++ ){ 			
    	for (j = 1 ; j <= n ; j++ ){
      		C[i*n+j] = 0.0;
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            C[i*n+j] += A[i*n+k] * B[k*n+j];
			}
		}
	}

}

void mat_mult_inv(int n,float *A, float *B,float *C){
	int i,j,k;
	for (i = 1 ; i <= n ; i++ ){ 			
    	for (j = 1 ; j <= n ; j++ ){
      		C[i*n+j] = 0.0;
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            C[i*n+j] += A[k*n+i] * B[k*n+j];
			}
		}
	}

}

void mult_eigenvec(int n,float *T,float *eigvec,float *mat_temp){
	int i,j,k;

	//#pragma acc loop
	for (i = 1 ; i <= n ; i++ ){ 								//eigenvec = eigenvec * T
    	for (j = 1 ; j <= n ; j++ ){
    		mat_temp[i*n+j] = 0;
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            mat_temp[i*n+j] += eigvec[k*n+i] * T[k*n+j];
			}
		}
	}

	//#pragma acc loop
	for (i = 1 ; i <= n ; i++ ){ 				
    	for (j = 1 ; j <= n ; j++ ){
        	//#pragma acc loop
        	for (k = 1 ; k <= n ; k++ ){
	            eigvec[i*n+j] = mat_temp[i*n+j] ;
			}
		}
	}
}

void new_T_mat(int max_i, int max_j,int n,float *mat,float *T){
	float tang, cose, sino;
	int c,r;

	tang = cal_tan(max_i,max_j,mat,n);
	cose = cal_cos(tang);
	sino = cal_sin(cose,tang);

	for (r = 1; r <= n; r++){				//Generate identity matrix
      	for (c = 1; c <= n; c++) 
     		T[r*n+c] = 0.0;		
      	T[r*n+r] = 1.0;	
	}
											//T Rotating matrix
    T[max_i*n+max_i] = cose;				
    T[max_j*n+max_j] = cose;
    T[max_i*n+max_j] = -sino; 				//Element to eliminate	
    T[max_j*n+max_i] = sino;		
    
}

__global__ void kernel_mat_mult_inv(int *n,float *A, float *B,float *C){

	float Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int e;

	//printf("n: %d\n", *n);

	if(row >= (*n) || col >= (*n)) 
		return;

	for (e = 1; e <= (*n); ++e)
		Cvalue += (A[e * (*n) + col]) * (B[e * (*n) + col]);
	
	C[row * (*n) + col] = Cvalue;
}

__global__ void kernel_mat_mult(int *n,float *A, float *B,float *C){

	float Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int e;

	//printf("n: %d\n", *n);

	if(row >= (*n) || col >= (*n)) 
		return;

	for (e = 1; e <= (*n); e++)
		Cvalue += (A[row * (*n) + e]) * (B[e * (*n) + col]);
	
	C[row * (*n) + col] = Cvalue;
}

void jacobiMultip (float *mat, int n, int ndm, float *eigvec, float eigval[],  int *nrot){

/*****************************+*******************************/	
	//On input
	//a: Contains the matrix to be diagonalized.
	//n: Order of matrix a.
	//ndim: Dimension of a.
	//eigvec: eigenvectors to be computed v

	//On output
	//eigval: Contains the eigenvalues in ascending order d
	//u: Contains the corresponding eigenvectors.
	//nrot: Number of Jacobi rotations.
/*****************************+*******************************/	

	int i,j;
	int *piv_elem;					//Keep coordenates of an elemnt i,j
	bool min = false;
	float EPS = .0000001;
	float *T;
	float *mat_temp; 

	int *d_n;
	float *d_mat; 
	float *d_T; 
	float *d_mat_temp; 

	size_t size = (n+1) * (n+1) * sizeof(float);

	hipError_t err;

	dim3 dimGrid( 512 );         // 512 x 1 x 1
	dim3 dimBlock( 1024, 1024 ); // 1024 x 1024 x 1 

	hipMalloc(&d_n,sizeof (int));
	hipMalloc(&d_mat, size);
	hipMalloc(&d_T, size);
	hipMalloc(&d_mat_temp, size);

	hipMemcpy(d_n, &n, sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat_temp, &mat_temp, size, hipMemcpyHostToDevice);

	//mat_temp = (float *) vector(1,n*n);					
	//T=(float *) vector(1,n*n);
	piv_elem=(int *) malloc (2 * sizeof (int));

	for (i = 1; i <= n; i++){						//Initializing Identity matrix
      	for (j = 1; j <= n; j++) 
     		eigvec[i*n+j] = 0.0;		
      	eigvec[i*n+i] = 1.0;	
	}
	//CPU
	for (*nrot = 0; min == false ; ++*nrot){
		max_elem(piv_elem,n,mat);	//Search for max element in tringle up

		if(fabs(mat[piv_elem[0]*n+piv_elem[1]]) < EPS || *nrot >= 100 ) //if max element doesnt exist more
			min=true;	
		
		else{
			new_T_mat(piv_elem[0],piv_elem[1],n,mat,T); //Calculate T matrix
			//mult_eigenvec(n,T,eigvec,mat_temp);							//Compute eigenvec
			mat_mult_inv(n,T,mat,mat_temp);
			mat_mult2(n,mat_temp,T,mat);
			
			// cudaMemcpy(d_mat, &mat, size, cudaMemcpyHostToDevice);
			// cudaMemcpy(d_T, &T, size, cudaMemcpyHostToDevice);
			// kernel_mat_mult_inv<<<dimGrid, dimBlock>>>(d_n,d_T,d_mat,d_mat_temp);
			// kernel_mat_mult<<<dimGrid, dimBlock>>>(d_n,d_mat_temp,d_T,d_mat);
			
			// cudaMemcpy(&mat, d_mat, size, cudaMemcpyDeviceToHost);

			
			//err=cudaMemcpy(&mat, d_mat, size, cudaMemcpyDeviceToHost);	
			//printf("Copy MAT off of device: %s\n",cudaGetErrorString(err));

			printf("\nRotación: %d\n",*nrot );
			for (i = 1; i <= n; ++i){
            printf("\n");
            for (j = 1; j <= n; ++j)
                printf("[%f]",mat[i*n+j]);
        }

		}

		for (i = 1; i <= n; ++i)
			eigval[i]=mat[i*n+i];

	}
 	hipFree(d_mat);
  	hipFree(d_T);
  	hipFree(d_mat_temp);

  	//free(mat_temp);

 	//*nrot = nrota;
 	//printf("rooooooot %d\n", *nrot);
	// free_vector(mat_temp,1,n*n);
	// free_vector(T,1,n);
	// free_vector(piv_elem,1,1);

}




int main(int argc, char **argv)
{
    int NP;
    char *nombreArchivo=argv[1];
    double S,E;
	int i, j, nrot=0;
    FILE *archivo;
    float *c;

    if (fopen(nombreArchivo, "r") == NULL){
        printf("File not found\n");
        return 1;
    }else{
        archivo = fopen(nombreArchivo, "r");
        fscanf(archivo, "%d", &NP);
        c =(float *)matrix(1,NP-1,1,NP-1);
        for (i = 0; i < NP; i++){
            for (j = 0; j < NP; j++){
                fscanf(archivo, "%f", &c[i*NP+j]);
            }
        }
        fclose(archivo);
    }
        float *d, *v, *e;
        
        d=(float *)vector(1,NP);
        v=(float *)matrix(1,NP,1,NP);
        e=(float *)convert_matrix(c,1,NP,1,NP);
        
        for (i = 1; i <= NP; ++i){
            printf("\n");
            for (j = 1; j <= NP; ++j)
                printf("[%f]",e[i*NP+j]);
        }

        printf("\n****** Finding Eigenvectors ******\n");
        //jacobi(e,NP,d,v,&nrot);
        get_walltime(&S);
              
        jacobiMultip(e,NP,NP,v,d,&nrot);
        
        get_walltime(&E);

        for (i = 1; i <= NP; ++i){
            printf("\n");
            for (j = 1; j <= NP; ++j)
                printf("[%f]",v[i*NP+j]);
        }
        
        printf("\nd\n");
        for (i = 1; i <= NP; ++i)
        {
            printf("[%f]",d[i]);
        }

        printf("\n******  Eigenvalues & Eigenvectors ******\n");
        for (i=1;i<=NP; i++) {
            printf("eigenvalue %3d, = %12.6f\n",i,d[i]);
            printf("eigenvector:\n");
            for (j=1;j<=NP; j++) {
                printf("%12.6f",v[i*NP+j]);
                if ((j % 5) == 0) printf("\n");
            }
                printf("\n");
        }

    printf("Rotations: %d\n",nrot );

    printf("Total time:%f sec\n", E-S);
	

    //free_vector(d,1,NP);
    //free_vector(v,1,NP*NP);
    //free_vector(e,1,NP*NP);
    // free(d);
    // free(v);
    // free(e);

    return 0;
	
}
