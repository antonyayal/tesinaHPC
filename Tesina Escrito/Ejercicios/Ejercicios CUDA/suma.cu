# include <hip/hip_runtime.h>
# include <stdio.h>
__global__ void add(int *a, int *b, int *c) {
     *c = *a + *b;
     printf("Suma device: %d\n",c);
}
int main( void) {
int a, b, c;                  // host copies
int *dev_a, *dev_b, *dev_c;   //device copies
int size = sizeof (int );     // space  for an integer

// allocate device copies of a, b, c
hipMalloc( (void **)&dev_a, size);
hipMalloc( (void **)&dev_b, size);
hipMalloc( (void **)&dev_c, size);

a = 2;
b = 7;

// copy inputs to device

hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
// lauch add() kernel on GPU, passing parameters
add<<<1,1>>> (dev_a, dev_b, dev_c);

// copy device results back to host copy of c

hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost);

printf("Suma cpu: %d\n",c);

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);

return 0;
}